#include "hip/hip_runtime.h"
#ifndef _ALPHA_FILTER_KERNEL_CU
#define _ALPHA_FILTER_KERNEL_CU

#include <opencv2/gpu/gpu.hpp>

#include "stdio.h"

__global__ void alphaKernel(cv::gpu::PtrStepSz<uchar> mat, double max_dist) 
{ 
	int x = threadIdx.x + blockIdx.x * blockDim.x;
	int y = threadIdx.y + blockIdx.y * blockDim.y;
	if (x < mat.cols && y < mat.rows) {
		double dist = sqrt(pow(abs((double) x - (double) mat.cols / 2), 2) + 
					   pow(abs((double) y - (double) mat.rows / 2), 2));
		dist /= max_dist;
		int alpha = 255;
		if (dist > 0.7) {
			alpha = (int) 255 - (dist-0.7)*1270;
			if (alpha < 0) {
				alpha = 0;
			}
		}
		mat.ptr(y)[4*x + 3] = alpha;
	}
} 

__global__ void subt(cv::gpu::PtrStepSz<uchar> mat, cv::gpu::PtrStepSz<uchar> out) 
{ 
	int x = threadIdx.x + blockIdx.x * blockDim.x;
	int y = threadIdx.y + blockIdx.y * blockDim.y;
	if (x < mat.cols && y < mat.rows) {
		out.ptr(y)[4*x] = 255 - mat.ptr(y)[4*x];
		out.ptr(y)[4*x + 1] = 255 - mat.ptr(y)[4*x + 1];
		out.ptr(y)[4*x + 2] = 255 - mat.ptr(y)[4*x + 2];
		out.ptr(y)[4*x + 3] = 255 - mat.ptr(y)[4*x + 3];
	}
} 

extern "C"
void alphaKernelCaller(cv::gpu::PtrStepSz<uchar> & x) 
{
	dim3 dimBlock(16, 16);
	dim3 dimGrid((x.cols + (dimBlock.x - 1)) / dimBlock.x, (x.rows + (dimBlock.y - 1)) / dimBlock.y);
	double maxDist = sqrt(pow((double) x.cols/2.0, 2) + pow((double) x.rows/2.0, 2));
   alphaKernel<<<dimGrid,dimBlock>>>(x, maxDist);
   hipDeviceSynchronize();
} 

extern "C"
void diff(cv::gpu::PtrStepSz<uchar> & x, cv::gpu::PtrStepSz<uchar> & out) {
	dim3 dimBlock(16, 16);
	dim3 dimGrid((x.cols + (dimBlock.x - 1)) / dimBlock.x, (x.rows + (dimBlock.y - 1)) / dimBlock.y);
    subt<<<dimGrid,dimBlock>>>(x, out);
   hipDeviceSynchronize();
}

#endif
